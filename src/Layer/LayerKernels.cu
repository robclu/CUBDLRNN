#include "hip/hip_runtime.h"
/*
 *  Layer header file for CUBDLRNN which defines the Cuda kernels that will 
 *  be used for the Layer class.
 *
 *  Copyright (C) 2015 Rob Clucas robclu1818@gmail.com
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published
 *  by the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT AN_size.y WARRANTY; without even the implied warranty of
 *  MERCHANTABILIT_size.y or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  _size.you should have received a copy of the GNU General Public License along
 *  with this program; if not, write to the Free Software Foundation,
 *	Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#ifndef LAYER_CUDA_KERNELS_INCLUDED
#define LAYER_CUDA_KERNELS_INCLUDED 

#include "../CudaHelp/Intellesense.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <>

namespace cubdlrnn {

	/* 
	 * ============================================================================
	 * Function		: MatrixSum
	 *
	 * Description  : Computes the sum of each row of a matrix in approx 
	 *                log_2(N)  where N = num elements in each row.
	 *
	 * Params       : matrix	: The matrix with the data for the sum
	 *              : index     : The index of the thread's element in the array
	 *              : N         : The number of elements in the row.
	 * ============================================================================
	 */
    // NOTE : FIX THIS
	template <class Type> 
	__device__ void MatrixSum( Type* matrix_in, int index, int N, Type* matrix_out ) {

		// Get the thread indexes
		int idx = blockIdx.x * blockDim.x + threadIdx.x;        
		int idy = blockIdx.y * blockDim.y + threadIdx.y;        
        int idz = blockIdx.z * blockDim.z + threadIdx.z;

	    while ( N > 1 ) {
			// If N is odd then the last element, the one with an odd index, 
			// gets added to the first element (so this adds comp time, but
			// allows all values of N to be handled  
			if ( N % 2 != 0 && idx == 0 ) {
				N -= 1;									        // Make the number of elements even 
				matrix_in[ index ] += matrix_in[ index + N ];
			}
			N /= 2;                                             // Halve the number of elements
			if ( idx < N ) {                                    // The first half threads add the value 
			    matrix_in[ index ] += matrix_in[ index + N ];   // of the element N positions ahead
			}
		    __syncthreads();                                    // Make sure all threads are finished 
		}
		if ( idx == 0 ) {                                       // Move result to output vector
			matrix_out[ idy * gridDim.z + idz ] = matrix_in[ index ];
		}
	}

	/*
	 * ============================================================================	 
	 * Function		: UpdateLayer
	 *
	 * Description  : Device kernel that updates the layer by computing all the
	 *                cell gate values, and computing the cell output.
	 *
	 * Params       : inputs	: The inputs are the new data, traditionally
	 *                            labelled x
	 *              : weights   : The weights are a 3D matrix where each page of
	 *                            the matrix is for the weights for an input type
	 *              : outputs   : The results of the update function
	 *
	 * NOTE         : This function should be called using many compute units
	 *                like would be done for a global kernel. But my GPU doesn't 
	 *                support  dynamic parallelism so this will have to wait.
	 *
	 *                Since shared memory is being used and there is only 16K of
	 *                that. This means 400 floats can be used. The Z dimension
	 *                will always be 4. The x dimension is the number of inputs
	 *                and the y dimension is the number of cells in the layer so 
	 *
	 *                num_cells x num_inputs < 1000
	 *
	 *                which should almost always be true.
	 * =============================================================================
	 */
	template<class Type, size_t max_input_size, size_t num_cells, size_t num_input_types>
	__global__ void UpdateLayer( Type* inputs, Type* weights, Type* outputs ) {     

		// Each gate has three wight matrices : 
		//     - For connections to data inputs
		//     - For connections to previous hidden state
		//     - For connections to cell states
		//
		// Each of these matrices are concatenated onto the end of the previous
		// one so that all data can be passed to the GPU in one pass. This can
		// be though of as a 3D matrix where each page of for one of the above
		// mentioned matrices this blockDim.z == 3 always (for now)

		int idx = blockIdx.x * blockDim.x + threadIdx.x;         // X index ( input index )
		int idy = blockIdx.y * blockDim.y + threadIdx.y;         // Y index ( cell index  )
        int idz = blockIdx.z * blockDim.z + threadIdx.z;         // Z index ( input type  )

		// Array (actually a flattened 3D matrix) for shared weights
		__shared__ Type shared_weights[ max_input_size   *       // Size of weights x dimension
			                            num_cells        *       // Size of weights y dimension 
									    num_input_types     ];   // Size of weights z dimension
		
		// Array (actually a flattened matrix) for shared inputs
		__shared__ Type shared_inputs[ num_input_types   *       // Size of inputs x dimension
			                           max_input_size       ];   // Size of inputs y dimension

		// Array (flattened 3D matrix) that holds the multiplication results
		__shared__ Type results_mult[ max_input_size  *
			                          num_cells       *
								      num_input_types     ];

		// Copy the weights into shared memory
		shared_weights[ ( idz * gridDim.x * gridDim.y ) +        // Offset due to pages (z)
			            ( idy * gridDim.x )             +        // Offset due to y
					    ( idx )                           ]      // Offset due to x 
	     = weights[     ( idz * gridDim.x * gridDim.y ) +      
			            ( idy * gridDim.x )             +          
					    ( idx )                           ];

		// The last thread along the y dimension must also 
		// also move an input into shared memory
		if ( idy == max_input_size ) {
			shared_inputs[ idy * num_input_types + idz ] = inputs[ idz * max_input_size + idy ];
		}

		// Make sure all threads are done
		__syncthreads();

		// Now do the multiplication and store the results in the 3D matrix (cube) of results 
		// Get the index of the element in the array
		int index = ( idz * gridDim.x * gridDim.y ) +            // Add offset due to z position
			        ( idy * gridDim.x )             +            // Add offset due to y position
					( idx );                                     // Add offset due to x position

		results_mult[ index ] = shared_weights[ index ] * 
			                    shared_inputs[ idx * num_input_types + idz ];

		// Make sure all threads are done
		__syncthreads();

		// When all the threads are done, there is a cube of dimension 
		// num_cells x num_inputs x num_input_types (3 for now) where each
		// element of the cube is the result of the weight input multiplication
		//
		// Since this is matrix multiplication, we need to do the addition along
		// each row of each page in the matrix. Call MatrixSum to do the
		// additions for the amtrix multiplication
		MatrixSum<Type>( &results_mult, index, max_input_size, outputs );
	}
}

#endif 
