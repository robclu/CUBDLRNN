/*
 *  Layer implementation file for CUBDLRNN that defines the member 
 *  implementation for the Layer Class 
 *
 *  Copyright (C) 2015 Rob Clucas robclu1818@gmail.com
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published
 *  by the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT AN_size.y WARRANTY; without even the implied warranty of
 *  MERCHANTABILIT_size.y or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  _size.you should have received a copy of the GNU General Public License along
 *  with this program; if not, write to the Free Software Foundation,
 *	Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "Layer.hpp"

// NOTE : For some reason if you don't use the namespace in this way 
//        nvcc will give errros
using namespace hipcubdlrnn::cell;

template< size_t num_inputs_x, size_t num_inputs_h, size_t num_cells,
	      size_t num_outputs , class Type > 
void Layer< num_inputs_x, num_inputs_h, num_cells, num_outputs, Type >
     ::Update( const Type* inputs_x, const Type* inputs_h ) {

	// NOTE : There should be a check here that determined is the GPU or CPU
	//        kernel should be used - which should actually be done when the 
	//        class is created
  	
	// Create a single array for the inputs
    Type* inputs[ maxInputs * NUM_INPUT_TYPES ];
	copy( inputs_x, inputs_x + num_inputs_x, inputs );             // Copy x inputs
	copy( inputs_h, inputs_h + num_inputs_h, inputs + maxInputs ); // Copy h inputs
	GetPreviousCellOutputs( inputs, 2 * maxInputs );               // Copy cell prev states

	// Create device inputs
	Type* inputs_d;
	hipMalloc( (void**)&inputs_d, maxInputs * NUM_INPUT_TYPES * sizeof( Type ) ); 
	hipMemcpy( inputs, inputs_d, maxInputs * NUM_INPUT_TYPES * sizeof( Type ),
			     hipMemcpyHostToDevice );

	// Create the output array
	Type* updated_results_h[ numCells * NUM_INPUT_TYPES ];
	Type* updated_results_d;
    hipMalloc( (void**)&updated_results_d, numCells * NUM_INPUT_TYPES * sizeof( Type ) );

	// Create weight matrix for device
	Type* weights_d;
	hipMalloc( (void**)&weights_d, numCells * maxInputs * NUM_INPUT_TYPES * sizeof( Type ) );
    hipMemcpy( Wi, weights_d, numCells * maxInputs * NUM_INPUT_TYPES * sizeof( Type ), 
			    hipMemcpyHostToDevice );	

	// Define the grid size 
	dim3 dimBlock( 1 );
	dim3 dimGrid( maxInputs, numCells, NUM_INPUT_TYPES );

	// Invoke the kernel 
	UpdateLayer<<< dimGrid, dimBlock >>>( inputs_d, weights_d, updated_results_d );

	// Get the results back from the GPU
	hipMemcpy( updated_results_h, updated_results_d, 
			    sizeof( updated_results_h ), hipMemcpyDeviceToHost );

	// Free memory
	hipFree( updated_results_d );
}

template< size_t num_inputs_x, size_t num_inputs_h, size_t num_cells, 
	      size_t num_outputs , class Type>
void Layer< num_inputs_x, num_inputs_h, num_cells, num_outputs , Type >
     ::GetPreviousCellOutputs( Type* input_array, size_t start_index  ) const {

		 for ( auto& cell : cells ) {
			 input_array[ start_index++ ] = cell.state_t;
		 }
}

